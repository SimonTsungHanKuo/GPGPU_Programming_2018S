#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
__global__ void
hello_kernel (char *odata, int num)
{
  char hello_str[12] = "Hello CUDA!";
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num)
    odata[idx] = hello_str[idx];
}

int
main (void)
{
  char *h_data, *d_data;
  const int strlen = 12;
  size_t strsize = strlen * sizeof (char);
  h_data = (char *) malloc (strsize);
  memset (h_data, 0, strlen);
  hipMalloc ((void **) &d_data, strsize);
  //cudaMemcpy (d_data, h_data, strsize, cudaMemcpyHostToDevice);
  int blocksize = 8;
  int nblock = strlen / blocksize + (strlen % blocksize == 0 ? 0 : 1);
  hello_kernel <<< nblock, blocksize >>> (d_data, strlen);
  hipMemcpy (h_data, d_data, sizeof (char) * strlen, hipMemcpyDeviceToHost);
  printf ("string:%s\n", h_data);
  free (h_data);
  hipFree (d_data);
}
